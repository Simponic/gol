#include "hip/hip_runtime.h"
#include "game.cuh"

// Count the number of life neighbors a cell has
__device__ int neighbors(struct GAME game, int x, int y) {
  int n = 0;

  for (int dy = -1; dy <= 1; dy++) {
    for (int dx = -1; dx <= 1; dx++) {
      if (!(dx == 0 && dy == 0) && (x+dx) > 0 && (y+dy) > 0 && (x+dx) < game.width+(game.padding*2) && (y+dy) < game.height+(game.padding*2)) {
        if (game.grid[(y+dy) * (game.width+game.padding*2) + (x+dx)]) {
          n++;
        }
      }
    }
  }
  return n;
}

// Compute the next iteration of a board
// We have to give it the newGrid as a parameter otherwise 
// each block will be computing its own version of the next grid
__global__ void next(struct GAME game, unsigned char* newGrid) {
  int idy = blockDim.y * blockIdx.y + threadIdx.y;
  int idx = blockDim.x * blockIdx.x + threadIdx.x;

  if (idy < game.height+game.padding*2 && idx < game.width+game.padding*2) {
    int my_neighbors = neighbors(game, idx, idy);
    int my_coord = idy * (game.width+game.padding*2) + idx;
    newGrid[my_coord] = 0; // It's possible that there are artifacts from the last iteration
    if (game.grid[my_coord]) {
      if (my_neighbors < 2 || my_neighbors > 3) {
        newGrid[my_coord] = 0;
      } else  {
        newGrid[my_coord] = 1;
      }
    } else {
      if (my_neighbors == 3) {
        newGrid[my_coord] = 1;
      }
    }
  }
}

// Randomly assign life value to each cell
void randomize(struct GAME* game) {
  for (int y = game->padding; y < game->height+game->padding; y++) {
    for (int x = game->padding; x < game->width+game->padding; x++) {
      game->grid[y*(game->width+game->padding*2) + x] = (unsigned char) rand() & 1;
    }
  }
}
