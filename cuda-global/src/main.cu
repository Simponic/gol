#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <time.h>
#include <stdio.h>
#include <string.h>
#include <cstring>

#include "file.cuh"
#include "game.cuh"
#include "create_grid.cuh"


/*
  Rules for life:
  Any live cell with fewer than two live neighbors dies (underpopulation).
  Any live cell with two or three live neighbors continues to live.
  Any live cell with more than three live neighbors dies (overpopulation).
  Any dead cell with exactly three live neighbors becomes a live cell (reproduction).
 */
#define BLOCK 32
#define PADDING 10
//#define VERBOSE 1
#define SEED 100

// gpuErrchk source: https://stackoverflow.com/questions/14038589/what-is-the-canonical-way-to-check-for-errors-using-the-cuda-runtime-api
#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort =
true) {
  if (code != hipSuccess) {
    fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code), file,
            line);
    if (abort)
      exit(code);
  }
}

// Do the simulation
void simulate(int argc, char** argv) {
  srand(SEED);
  hipEvent_t global_start, global_end;
  hipEventCreate(&global_start);
  hipEventCreate(&global_end);
  hipEventRecord(global_start);
  char* filename;
  struct GAME game;
  game.padding = PADDING;
  int iterations, log_each_step;
  if (argc == 7) {
    // Parse the arguments
    filename = argv[2];
    game.width = atoi(argv[3]);
    game.height = atoi(argv[4]);
    iterations = atoi(argv[5]);
    log_each_step = atoi(argv[6]);
  } else {
    printf("Usage: ./gol simulate <filename | random> <width> <height> <iterations> <log-each-step?1:0>\n");
    filename = "random";
    game.height = 10;
    game.width = 10;
    iterations = 5;
    log_each_step = 0;
  }

  // Allocate space for current grid (1 byte per tile)
  int size = (game.height+(2*game.padding)) * (game.width+(2*game.padding)) * sizeof(unsigned char);
  game.grid = (unsigned char*)malloc(size);
  memset(game.grid, 0, size);

  // Choose where to read initial position
  if (strcmp(filename, "random") == 0) {
    randomize(&game);
  } else {
    read_in(filename, &game);
  }  

  char iteration_file[1024];

  // Allocate device memory
  unsigned char* grid_d;
  unsigned char* newGrid;
  gpuErrchk(hipMalloc(&grid_d, size));
  gpuErrchk(hipMalloc(&newGrid, size));
  gpuErrchk(hipMemcpy(grid_d, game.grid, size, hipMemcpyHostToDevice)); // Copy the initial grid to the device
  free(game.grid);
  game.grid = grid_d; // Use the device copy

  // The grid that we will copy results 
  unsigned char* grid_h = (unsigned char*)malloc(size);
  unsigned char* temp;

  // Calculate grid width for kernel
  int grid_width = (int)ceil((game.width+(2*game.padding))/(float)BLOCK);
  int grid_height = (int)ceil((game.height+(2*game.padding))/(float)BLOCK);
  dim3 dim_grid(grid_width, grid_height, 1);
  dim3 dim_block(BLOCK, BLOCK, 1);

  // Timing
  hipEvent_t start, end;
  hipEventCreate(&start);
  hipEventCreate(&end);
  double time_computing_life = 0;
  float local_time = 0;

  for (int i = 0; i <= iterations; i++) {
    // Iteration 0 will just be the initial grid
    if (i > 0) {
      hipEventRecord(start);
      // Compute the next grid
      next<<<dim_grid, dim_block>>>(game, newGrid);
      hipEventRecord(end);
      hipEventSynchronize(end);
      hipEventElapsedTime(&local_time, start, end);
      time_computing_life += local_time/1000;

      // Swap game.grid and newGrid
      temp = game.grid;
      game.grid = newGrid;
      newGrid = temp;
    }
    if (log_each_step) {
      // If we are logging each step, perform IO operations
      gpuErrchk(hipMemcpy(grid_h, game.grid, size, hipMemcpyDeviceToHost));
      #ifdef VERBOSE
        // Print the board without the padding elements
        printf("\n===Iteration %i===\n", i);
        for (int y = game.padding; y < game.height+game.padding; y++) {
          for (int x = game.padding; x < game.width+game.padding; x++) {
            printf("%s ", grid_h[y*(game.width+2*game.padding) + x] ? "X" : " ");
          }
          printf("\n");
        }
        printf("===End iteration %i===\n", i);
      #endif
      // Save to a file
      sprintf(iteration_file, "output/iteration-%07d.bin", i);
      temp = game.grid;
      game.grid = grid_h;
      write_out(iteration_file, &game);
      game.grid = temp;
    }
  }
  hipEventRecord(global_end);
  hipEventSynchronize(global_end);
  float global_time;
  hipEventElapsedTime(&global_time, global_start, global_end);

  printf("\n===Timing===\nTime computing life: %f\nClock time: %f\n", time_computing_life, global_time/(double)1000);
}

int main(int argc, char** argv) {
  if (argc >= 2) {
    if (strcmp(argv[1], "simulate") == 0) {
      simulate(argc, argv);
    } else if (strcmp(argv[1], "create-grid") == 0) {
      create_grid(argc, argv);
    } else {
      printf("Unknown input: %s\n", argv[1]);
      exit(1);
    }
  } else {
    printf("Usage: ./gol <simulate | create-grid>\n");
    exit(1);
  }
  return 0;
}
